#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

float* loadCSV(const char* filename, int* numElements) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        perror("Failed to open file");
        return NULL;
    }

    int capacity = 1024;
    float* data = (float*)malloc(sizeof(float) * capacity);
    if (!data) {
        perror("malloc failed");
        fclose(file);
        return NULL;
    }

    int count = 0;
    while (fscanf(file, "%f,", &data[count]) == 1) {
        count++;
        if (count >= capacity) {
            capacity *= 2;
            float* temp = (float*)realloc(data, sizeof(float) * capacity);
            if (!temp) {
                perror("realloc failed");
                free(data);
                fclose(file);
                return NULL;
            }
            data = temp;
        }
    }

    fclose(file);
    *numElements = count;
    return data;
}

int main() {
    float* hostData;
    int numElements;

    hostData = loadCSV("Downloads/x_train.csv", &numElements);
    if (!hostData) return 1;

    // Print first 10 loaded values
    for (int i = 0; i < 10 && i < numElements; i++) {
        printf("Value %d: %f\n", i, hostData[i]);
    }

    float* deviceData;
    hipError_t err;

    err = hipMalloc((void**)&deviceData, numElements * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        free(hostData);
        return 1;
    }

    err = hipMemcpy(deviceData, hostData, numElements * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(err));
        hipFree(deviceData);
        free(hostData);
        return 1;
    }

    // No kernel yet — just testing load and transfer

    hipFree(deviceData);
    free(hostData);

    return 0;
}
